#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h> 
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hiprand/hiprand_kernel.h>

#include "main.h"
#include "gpu.h"
#include "init.h"


/*-----------------------------------------------------------------------------------------*/
/* Define dynamic arrays and variables of GPU                                              */
/*-----------------------------------------------------------------------------------------*/
T_real *GPU_dataT;       // Array for the transposed matrix of data instances
T_real *GPU_centroid;    // Array for the matrix of centroids
T_real *GPU_centroidT;   // Array for the transposed matrix of centroids
T_real *GPU_package;     // Array for the packages used in UpdateCentroids
int *GPU_label;          // Array for cluster labels of data points
int *GPU_count;          // Count of data instances in each cluster
__device__ unsigned long long int GPU_track_sum;  // Sum of label changes in two consecutive iterations
unsigned long long int *AdrGPU_track_sum = NULL;  // Address of GPU_track_sum
hiprandState *devStates;                           // States for using cuRAND library

hipblasHandle_t cublasHandle;                      // Handle for cuBLAS library

hipEvent_t start;                                // CUDA event used for time measurement
hipEvent_t stop;


/*-----------------------------------------------------------------------------------------*/
/* Init and finalize the GPU device                                                        */
/*-----------------------------------------------------------------------------------------*/
void gpuInit(void)
{
    hipInit(0);

    // Allocate memory space for the dynamic arrays
    CHECK_CUDA_SUCCESS(hipMalloc((void**) &GPU_dataT, sizeof(T_real)*NbDims*NbPoints), "Dynamic allocation for GPU_dataT");
    CHECK_CUDA_SUCCESS(hipMalloc((void**) &GPU_centroid, sizeof(T_real)*NbClusters*NbDims), "Dynamic allocation for GPU_centroid");
    CHECK_CUDA_SUCCESS(hipMalloc((void**) &GPU_centroidT, sizeof(T_real)*NbDims*NbClusters), "Dynamic allocation for GPU_centroidT");
    CHECK_CUDA_SUCCESS(hipMalloc((void**) &GPU_package, sizeof(T_real)*NbDims*NbClusters*NbPackages), "Dynamic allocation for GPU_package");
    CHECK_CUDA_SUCCESS(hipMalloc((void**) &GPU_label, sizeof(int)*NbPoints), "Dynamic allocation for GPU_label");
    CHECK_CUDA_SUCCESS(hipMalloc((void**) &GPU_count, sizeof(int)*NbClusters), "Dynamic allocation for GPU_count");
	CHECK_CUDA_SUCCESS(hipMalloc((void**) &devStates, sizeof(hiprandState)*NbClusters), "Dynamic allocation for devStates");
    CHECK_CUDA_SUCCESS(hipGetSymbolAddress((void **) &AdrGPU_track_sum, GPU_track_sum), "Get the address of GPU_track_sum");

    // Turn CPU arrays dataT, centroid and label into "pinned" memory areas
    CHECK_CUDA_SUCCESS(hipHostRegister(dataT, sizeof(T_real)*NbDims*NbPoints, hipHostRegisterPortable), "Turn dataT into pinned memory");
    CHECK_CUDA_SUCCESS(hipHostRegister(label, sizeof(int)*NbPoints, hipHostRegisterPortable), "Turn label into pinned memory");
    CHECK_CUDA_SUCCESS(hipHostRegister(centroid, sizeof(T_real)*NbClusters*NbDims, hipHostRegisterPortable), "Turn centroid into pinned memory");
    CHECK_CUDA_SUCCESS(hipHostRegister(&track, sizeof(unsigned long long int), hipHostRegisterPortable), "Turn track into pinned memory");

    // Initialize CUBLAS lib usage
    CHECK_CUBLAS_SUCCESS(hipblasCreate(&cublasHandle), "Init of the CUBLAS lib handle"); 

    // Create events
    CHECK_CUDA_SUCCESS(hipEventCreateWithFlags(&start, hipEventBlockingSync), "Create the event start");
    CHECK_CUDA_SUCCESS(hipEventCreateWithFlags(&stop, hipEventBlockingSync), "Create the event stop");
}


void gpuFinalize(void)
{
    // Free dynamic allocations on GPU
    CHECK_CUDA_SUCCESS(hipFree(GPU_dataT), "Free the dynamic allocation for GPU_dataT");
    CHECK_CUDA_SUCCESS(hipFree(GPU_centroid), "Free the dynamic allocation for GPU_centroid");
    CHECK_CUDA_SUCCESS(hipFree(GPU_centroidT), "Free the dynamic allocation for GPU_centroidT");
    CHECK_CUDA_SUCCESS(hipFree(GPU_package), "Free the dynamic allocation for GPU_package");
    CHECK_CUDA_SUCCESS(hipFree(GPU_label), "Free the dynamic allocation for GPU_label");
    CHECK_CUDA_SUCCESS(hipFree(GPU_count), "Free the dynamic allocation for GPU_count");
    CHECK_CUDA_SUCCESS(hipFree(devStates), "Free the dynamic allocation for devStates");

    // Turn "pinned" CPU arrays into std array
    CHECK_CUDA_SUCCESS(hipHostUnregister(dataT), "Turn pinned dataT into standard array");
    CHECK_CUDA_SUCCESS(hipHostUnregister(label), "Turn pinned label into standard array");
    CHECK_CUDA_SUCCESS(hipHostUnregister(centroid), "Turn pinned centroid into standard array");
    CHECK_CUDA_SUCCESS(hipHostUnregister(&track), "Turn pinned track into standard array");

    // Destroy events
    CHECK_CUDA_SUCCESS(hipEventDestroy(start), "Destroy the event start");
    CHECK_CUDA_SUCCESS(hipEventDestroy(stop), "Destroy the event stop");

    // Free CUBLAS lib usage
    CHECK_CUBLAS_SUCCESS(hipblasDestroy(cublasHandle), "Free the CUBLAS lib");
}


/*-----------------------------------------------------------------------------------------*/
/* Transfer of CPU input data into GPU symbols                                             */
/*-----------------------------------------------------------------------------------------*/
void gpuSetDataOnGPU(void)
{
    CHECK_CUDA_SUCCESS(hipMemcpy(GPU_dataT, dataT, sizeof(T_real)*NbDims*NbPoints, hipMemcpyHostToDevice),
                       "Transfer dataT --> GPU_dataT");
    if (INPUT_INITIAL_CENTROIDS != "") {
        CHECK_CUDA_SUCCESS(hipMemcpy(GPU_centroid, centroid, sizeof(T_real)*NbClusters*NbDims, hipMemcpyHostToDevice),
                       "Transfer centroid --> GPU_centroid");
    }
}


/*-----------------------------------------------------------------------------------------*/
/* Transfer of GPU results into CPU array                                                  */
/*-----------------------------------------------------------------------------------------*/
void gpuGetResultOnCPU(void)
{
    CHECK_CUDA_SUCCESS(hipMemcpy(label, GPU_label, sizeof(int)*NbPoints, hipMemcpyDeviceToHost),
                       "Transfer GPU_label-->label");
    CHECK_CUDA_SUCCESS(hipMemcpy(centroid, GPU_centroid, sizeof(T_real)*NbClusters*NbDims, hipMemcpyDeviceToHost),
                       "Transfer GPU_centroid-->centroid");
}


/*-----------------------------------------------------------------------------------------*/
/* Select initial centroids                                                                */
/*-----------------------------------------------------------------------------------------*/
__global__ void SetupcuRand(hiprandState *state)
{
    int col = threadIdx.x + blockIdx.x * blockDim.x;

    if (col < NbClusters) 
        hiprand_init(0, col, 0, &state[col]);
}


__global__ void InitializeCentroids(hiprandState *state, T_real *GPU_centroidT, T_real *GPU_dataT)
{
    int col = threadIdx.x + blockIdx.x * blockDim.x;

    if (col < NbClusters) {
        hiprandState localState = state[col];
        int idx = (ceil(NbPoints * hiprand_uniform(&localState))) - 1;  // Control idx in [0, NbPoints - 1]
        for (int j = 0; j < NbDims; j++)
            GPU_centroidT[j*NbClusters + col] = GPU_dataT[j*NbPoints + idx];
    }
}


/*-----------------------------------------------------------------------------------------*/
/* Compute point-centroid distances and assign each point to a cluter                      */
/*-----------------------------------------------------------------------------------------*/
__global__ void ComputeAssign(T_real *GPU_dataT, T_real *GPU_centroid, int *GPU_label, unsigned long long int *AdrGPU_track_sum)
{
    int idx = blockIdx.x * BSXN + threadIdx.x;
    __shared__ unsigned long long int shTrack[BSXN];
    shTrack[threadIdx.x] = 0;

    if (idx < NbPoints) {
        int min = 0;
        T_real diff, dist_sq, minDist_sq;
        for (int k = 0; k < NbClusters; k++) {
            dist_sq = 0.0f;
            // Calculate the square of distance between instance idx and centroid k
            for(int j = 0; j < NbDims; j++) {
                diff = (GPU_dataT[j*NbPoints + idx] - GPU_centroid[k*NbDims + j]);
                dist_sq += (diff*diff);
            }
            // Find and record the nearest centroid to instance idx
            if (dist_sq < minDist_sq || k == 0) {
                minDist_sq = dist_sq;
                min = k;
            }
        }
        // Change the label if necessary
        if (GPU_label[idx] != min) {
            shTrack[threadIdx.x] = 1;
            GPU_label[idx] = min;
        }
    }

    // Count the changes of label into "track": two-part reduction
    // 1 - Parallel reduction of 1D block shared array shTrack[*] into shTrack[0],
    //     kill useless threads step by step, only thread 0 survives at the end.
    #if BSXN > 512
        __syncthreads();
        if (threadIdx.x < 512)
            shTrack[threadIdx.x] += shTrack[threadIdx.x + 512];
        else
            return;
    #endif

    #if BSXN > 256
        __syncthreads();
        if (threadIdx.x < 256)
            shTrack[threadIdx.x] += shTrack[threadIdx.x + 256];
        else
            return;
    #endif

    #if BSXN > 128
        __syncthreads();
        if (threadIdx.x < 128)
            shTrack[threadIdx.x] += shTrack[threadIdx.x + 128];
        else
            return;
    #endif

    #if BSXN > 64
        __syncthreads();
        if (threadIdx.x < 64)
            shTrack[threadIdx.x] += shTrack[threadIdx.x + 64];
        else
            return;
    #endif

    #if BSXN > 32
        __syncthreads();
        if (threadIdx.x < 32)
            shTrack[threadIdx.x] += shTrack[threadIdx.x + 32];
        else
            return;
    #endif

    #if BSXN > 16
        __syncwarp();          // avoid races between threads within the same warp
        if (threadIdx.x < 16)
            shTrack[threadIdx.x] += shTrack[threadIdx.x + 16];
        else
            return;
    #endif

    #if BSXN > 8
        __syncwarp();          // avoid races between threads within the same warp
        if (threadIdx.x < 8)
            shTrack[threadIdx.x] += shTrack[threadIdx.x + 8];
        else
            return;
    #endif

    #if BSXN > 4
        __syncwarp();          // avoid races between threads within the same warp
        if (threadIdx.x < 4)
            shTrack[threadIdx.x] += shTrack[threadIdx.x + 4];
        else
            return;
    #endif

    #if BSXN > 2
        __syncwarp();          // avoid races between threads within the same warp
        if (threadIdx.x < 2)
            shTrack[threadIdx.x] += shTrack[threadIdx.x + 2];
        else
            return;
    #endif

    #if BSXN > 1
        __syncwarp();          // avoid races between threads within the same warp
        if (threadIdx.x < 1)
            shTrack[threadIdx.x] += shTrack[threadIdx.x + 1];
        else
            return;
    #endif

    // 2 - Final reduction into a global array
    if (shTrack[0] > 0)
        atomicAdd(AdrGPU_track_sum, shTrack[0]);
}


/*-----------------------------------------------------------------------------------------*/
/* Update centroids												                           */
/*-----------------------------------------------------------------------------------------*/
__global__ void UpdateCentroids_Step1_Child(int pid, int offset, int length, int *GPU_label, T_real *GPU_package, T_real *GPU_dataT, int *GPU_count)
{
    __shared__ T_real shTabV[BSYD][BSXP];            // Tab of instance values
    __shared__ int shTabL[BSXP];                     // Tab of labels (cluster Id)
    // Index initialization
    int baseRow = blockIdx.y * BSYD;                 // Base row of the block
    int row = baseRow + threadIdx.y;                 // Row of child thread
    int baseCol = blockIdx.x * BSXP + offset;        // Base column of the block
    int col = baseCol + threadIdx.x;                 // Column of child thread
    int cltIdx = threadIdx.y * BSXP + threadIdx.x;   // 1D cluster index

    // Load the values and cluster labels of instances into sh mem tables
    if (col < (offset + length) && row < NbDims) {
        shTabV[threadIdx.y][threadIdx.x] = GPU_dataT[row*NbPoints + col];
        if (threadIdx.y == 0)
            shTabL[threadIdx.x] = GPU_label[col];
    }

    __syncthreads();                 // Wait for all data loaded into the sh mem

    // Compute partial evolution of centroid related to cluster number 'cltIdx'
    if (cltIdx < NbClusters) {             // Required condition: NbClusters <= BSXP*BSYD <= 1024
        #define BlND (NbDims < BSYD ? NbDims : BSYD) // BlND: nb of dims stored by block
        T_real Sv[BlND];             // Sum of values in BlND dimensions
        for (int j = 0; j < BlND; j++)
            Sv[j] = 0.0f;            // Init the tab Sv to zeros
        int count = 0;               // Init the counter of instances

        // - Accumulate contributions to cluster number 'cltIdx'
        for (int x = 0; x < BSXP && (baseCol + x) < (offset + length); x++) {
            if (shTabL[x] == cltIdx) { 
                count++;
                for (int y = 0; y < BSYD && (baseRow + y) < NbDims; y++)
                    Sv[y] += shTabV[y][x];
            }
        }

        // - Save the contribution of block into global contribution of the package
        if (count != 0) {
            if (blockIdx.y == 0)
                atomicAdd(&GPU_count[cltIdx], count);
            int dmax = (blockIdx.y == NbDims/BSYD ? NbDims%BSYD : BSYD);
            for (int j = 0; j < dmax; j++)  // BlND_max: nb of dims managed by blk
                atomicAdd(&GPU_package[(baseRow + j)*NbClusters*NbPackages + NbClusters*pid + cltIdx], Sv[j]);
        }
    } 
}


__global__ void UpdateCentroids_Step1_Parent(int *GPU_label, T_real *GPU_package, T_real *GPU_dataT, int *GPU_count)
{
    int tid = threadIdx.x;              // Thread id

    if (tid < NbPackages) {
        int offset, length, quotient, remainder;
        int np = NbPackages/nStreams1 + (NbPackages%nStreams1 > 0 ? 1 : 0);  // Nb of packages for each stream
        int pid;                        // Id of package
        hipStream_t stream;
        dim3 Dg, Db;

        hipStreamCreateWithFlags(&stream, hipStreamDefault); 

	    quotient = NbPoints/NbPackages;
        remainder = NbPoints%NbPackages;

        Db.x = BSXP;
        Db.y = BSYD;
        Db.z = 1;

        Dg.y = NbDims/Db.y + (NbDims%Db.y > 0 ? 1 : 0);
        Dg.z = 1;

        for (int i = 0; i < np; i++) {
            pid = i*nStreams1 + tid;     // Calculate the id of package
            if (pid < NbPackages) {
                offset = (pid < remainder ? ((quotient + 1) * pid) : (quotient * pid + remainder));
                length = (pid < remainder ? (quotient + 1) : quotient);
                Dg.x = length/Db.x + (length%Db.x > 0 ? 1 : 0);
                // Launch a child kernel on a stream to process a package
                UpdateCentroids_Step1_Child<<<Dg,Db,0,stream>>>(pid, offset, length, GPU_label, GPU_package, GPU_dataT, GPU_count);
            }
        }
        hipStreamDestroy(stream); 
    }
}


__global__ void UpdateCentroids_Step2_Child(int pid, T_real *GPU_package, T_real *GPU_centroidT, int *GPU_count)
{
    int rowC = blockIdx.y;                           // Row of child thread
    int colC = blockIdx.x * BSXK + threadIdx.x;      // Col of child thread

    if (colC < NbClusters && rowC < NbDims)
        if (GPU_count[colC] != 0)
            atomicAdd(&GPU_centroidT[rowC*NbClusters + colC], GPU_package[rowC*NbClusters*NbPackages + NbClusters*pid + colC] / GPU_count[colC]);
}


__global__ void UpdateCentroids_Step2_Parent(T_real *GPU_package, T_real *GPU_centroidT, int *GPU_count)
{
    int tid = threadIdx.x;

    if (tid < NbPackages) {
        int np = NbPackages/nStreams2 + (NbPackages%nStreams2 > 0 ? 1 : 0); // Nb of packages for each stream
        int pid;   // Id of package
        hipStream_t stream;
        dim3 Dg, Db;

        hipStreamCreateWithFlags(&stream, hipStreamDefault); 

        Db.x = BSXK;
        Db.y = 1;
        Db.z = 1;
        Dg.x = NbClusters/BSXK + (NbClusters%BSXK > 0 ? 1 : 0);
        Dg.y = NbDims;
        Dg.z = 1;

        for (int i = 0; i < np; i++) {
            pid = i*nStreams2 + tid;   // Calculate the id of package
            if (pid < NbPackages) 
                UpdateCentroids_Step2_Child<<<Dg,Db,0,stream>>>(pid, GPU_package, GPU_centroidT, GPU_count);
        }
        hipStreamDestroy(stream); 
    }
}


/*-----------------------------------------------------------------------------------------*/
/* K-means clustering on the GPU                                                           */
/*-----------------------------------------------------------------------------------------*/
void gpuKmeans(void)
{
    dim3 Dg, Db;
    double tolerance = 0.0;
    float elapsed;
    T_real alpha, beta;       // Parameters for CUBLAS_GEAM

    // Reset global variables to zeros
    NbIters = 0;              
    Tms_init = 0.0f;
    Tms_transpose = 0.0f;
    Tms_compute_assign = 0.0f;
    Tms_update = 0.0f;

    if (INPUT_INITIAL_CENTROIDS != "") {
        alpha = 1.0f;
        beta = 0.0f;
        // Get GPU_centroidT by transposing GPU_centroid
        CHECK_CUDA_SUCCESS(hipEventRecord(start, 0), "Record the beginning time of transposing GPU_centroid");
        CHECK_CUBLAS_SUCCESS(CUBLAS_GEAM(cublasHandle, 
                             HIPBLAS_OP_T,
                             HIPBLAS_OP_N,
                             NbClusters, NbDims,
                             &alpha,
                             GPU_centroid, NbDims,
                             &beta,
                             NULL, NbClusters,
                             GPU_centroidT, NbClusters), "Use CUBLAS_GEAM to transpose GPU_centroid"); 
        CHECK_CUDA_SUCCESS(hipEventRecord(stop, 0), "Record the ending time of transposing GPU_centroid");
        CHECK_CUDA_SUCCESS(hipEventSynchronize(stop), "Wait for completion of transposing GPU_centroid");
        CHECK_CUDA_SUCCESS(hipEventElapsedTime(&elapsed, start, stop), "Elapsed time of transposing GPU_centroid");
        Tms_transpose += elapsed;
    } else {
        // Select initial centroids randomly
        Db.x = BSXK;
        Db.y = 1;
        Db.z = 1;
        Dg.x = NbClusters/Db.x + (NbClusters%Db.x > 0 ? 1 : 0);
        Dg.y = 1;
        Dg.z = 1;
        CHECK_CUDA_SUCCESS(hipEventRecord(start, 0), "Record the beginning time of initialization");
        SetupcuRand<<<Dg,Db>>>(devStates);
        InitializeCentroids<<<Dg,Db>>>(devStates, GPU_centroidT, GPU_dataT);
        CHECK_CUDA_SUCCESS(hipEventRecord(stop, 0), "Record the ending time of initialization");
        CHECK_CUDA_SUCCESS(hipEventSynchronize(stop), "Wait for completion of initialization");
        CHECK_CUDA_SUCCESS(hipEventElapsedTime(&elapsed, start, stop), "Elapsed time of initialization");
        Tms_init += elapsed;

        // Get GPU_centroid by transposing GPU_centroidT
        alpha = 1.0f;
        beta = 0.0f;
        CHECK_CUDA_SUCCESS(hipEventRecord(start, 0), "Record the beginning time of transposing GPU_centroidT");
        CHECK_CUBLAS_SUCCESS(CUBLAS_GEAM(cublasHandle,
                             HIPBLAS_OP_T,
                             HIPBLAS_OP_N,
                             NbDims, NbClusters,
                             &alpha,
                             GPU_centroidT, NbClusters,
                             &beta,
                             NULL, NbDims,
                             GPU_centroid, NbDims), "Use CUBLAS_GEAM to transpose GPU_centroidT");
        CHECK_CUDA_SUCCESS(hipEventRecord(stop, 0), "Record the ending time of transposing GPU_centroidT");
        CHECK_CUDA_SUCCESS(hipEventSynchronize(stop), "Wait for completion of transposing GPU_centroidT");
        CHECK_CUDA_SUCCESS(hipEventElapsedTime(&elapsed, start, stop), "Elapsed time of transposing GPU_centroidT");
        Tms_transpose += elapsed;
    }

    // CHECK_CUDA_SUCCESS(hipMemset(GPU_label, 0, sizeof(int)*NbPoints), "Reset GPU_label to zeros");

    do {
        // Compute point-centroid distances & Assign each point to its nearest centroid
        Db.x = BSXN;
        Db.y = 1;
        Db.z = 1;
        Dg.x = NbPoints/Db.x + (NbPoints%Db.x > 0 ? 1 : 0);
        Dg.y = 1;
        Dg.z = 1;
        CHECK_CUDA_SUCCESS(hipEventRecord(start, 0), "Record the beginning time of ComputeAssign");
        CHECK_CUDA_SUCCESS(hipMemset(AdrGPU_track_sum, 0, sizeof(unsigned long long int)*1), "Reset GPU_track_sum to zero");
        ComputeAssign<<<Dg,Db>>>(GPU_dataT, GPU_centroid, GPU_label, AdrGPU_track_sum);
        CHECK_CUDA_SUCCESS(hipMemcpy(&track, AdrGPU_track_sum, sizeof(unsigned long long int)*1, hipMemcpyDeviceToHost),
                           "Transfer GPU_track_sum-->track");
        CHECK_CUDA_SUCCESS(hipEventRecord(stop, 0), "Record the ending time of ComputeAssign");
        CHECK_CUDA_SUCCESS(hipEventSynchronize(stop), "Wait for completion of ComputeAssign");
        CHECK_CUDA_SUCCESS(hipEventElapsedTime(&elapsed, start, stop), "Elapsed time of ComputeAssign");
        Tms_compute_assign += elapsed;

        // Update centroids - Step1
        CHECK_CUDA_SUCCESS(hipEventRecord(start, 0), "Record the beginning time of Update Step1");
        CHECK_CUDA_SUCCESS(hipMemset(GPU_count, 0, sizeof(int)*NbClusters), "Reset GPU_count to zeros");
        CHECK_CUDA_SUCCESS(hipMemset(GPU_package, 0, sizeof(T_real)*NbDims*NbClusters*NbPackages), "Reset GPU_package to zeros");
        CHECK_CUDA_SUCCESS(hipMemset(GPU_centroidT, 0, sizeof(T_real)*NbDims*NbClusters), "Reset GPU_centroidT to zeros");
        UpdateCentroids_Step1_Parent<<<1,nStreams1>>>(GPU_label, GPU_package, GPU_dataT, GPU_count);
        CHECK_CUDA_SUCCESS(hipEventRecord(stop, 0), "Record the ending time of Update Step1");
        CHECK_CUDA_SUCCESS(hipEventSynchronize(stop), "Wait for completion of Update Step1");
        CHECK_CUDA_SUCCESS(hipEventElapsedTime(&elapsed, start, stop), "Elapsed time of Update Step1");
        Tms_update += elapsed;
		
        // Update centroids - Step2
        alpha = 1.0f;
        beta = 0.0f;
        CHECK_CUDA_SUCCESS(hipEventRecord(start, 0), "Record the beginning time of Update Step2");
        UpdateCentroids_Step2_Parent<<<1,nStreams2>>>(GPU_package, GPU_centroidT, GPU_count);
        CHECK_CUBLAS_SUCCESS(CUBLAS_GEAM(cublasHandle,
                             HIPBLAS_OP_T,
                             HIPBLAS_OP_N,
                             NbDims, NbClusters,
                             &alpha,
                             GPU_centroidT, NbClusters,
                             &beta,
                             NULL, NbDims,
                             GPU_centroid, NbDims), "Use CUBLAS_GEAM to transpose GPU_centroidT");
        CHECK_CUDA_SUCCESS(hipEventRecord(stop, 0), "Record the ending time of Update Step2");
        CHECK_CUDA_SUCCESS(hipEventSynchronize(stop), "Wait for completion of Update Step2");
        CHECK_CUDA_SUCCESS(hipEventElapsedTime(&elapsed, start, stop), "Elapsed time of Update Step2");
        Tms_update += elapsed;

        // Calculate the variables for checking stopping criteria
        NbIters++;   // Count the number of iterations
        tolerance = (double)track / NbPoints;     
        //printf("Track = %llu  Tolerance = %lf\n", track, tolerance); 
    } while (NbIters < MaxNbIters);
}
