#include "hip/hip_runtime.h"
#include <iostream>
#include <random>
#include <algorithm>
#include <cfloat>
#include <chrono>
#include "./device.cuh"

__device__ float squared_l2_distance(float x_1, float x_2) {
    float a = x_1 - x_2;
    return a*a;
}


__global__ void assign_clusters(int attrs_size, int k, int dims,
    float* __restrict__ attrs, float* __restrict__ mean, float* __restrict__ sum,
    unsigned int* __restrict__ counts, int* __restrict__ assigments)
{ 
    const int global_index = blockIdx.x * blockDim.x + threadIdx.x;

    float best_distance{FLT_MAX};
    int best_cluster{-1};
    float distance{0};
    for (int cluster = 0; cluster < k; ++cluster) {
        for(int d{0}; d < dims; d++)
            distance += squared_l2_distance(attrs[(d * attrs_size) + global_index], mean[(cluster * dims) + d]);
        
        if (distance < best_distance) {
            best_distance = distance;
            best_cluster = cluster;
        }
        distance = 0;
    }
    assigments[global_index] = best_cluster;

    int val_id{0}, sum_id{0};
    for(int cluster{0}; cluster < k; cluster++) {
        for(int d{0}; d < dims; d++) {
            val_id       = (d * attrs_size) + global_index;
            sum_id       = attrs_size * cluster * dims + val_id;
            sum[sum_id]  = (best_cluster == cluster) ? attrs[val_id] : 0;
        }
        counts[attrs_size * cluster + global_index] = (best_cluster == cluster) ? 1 : 0;
    }
}


__global__ void tree_reduction(int CUs, int attrs_per_CU, int attrs_size, int remaining_attrs,
    int k, int dims, float* __restrict__ sums, unsigned int* __restrict__ counts)
{
    extern __shared__ float shared_sum[];
    unsigned int* shared_count = (unsigned int*) &shared_sum[CUs];

    const int cluster        = blockIdx.y * blockDim.y + threadIdx.y;
    const int d              = blockIdx.z * blockDim.z + threadIdx.z;
    const int local_idx      = threadIdx.x;
    const int attr_start_idx = attrs_per_CU * local_idx;
    const int n_attrs        = (local_idx == CUs-1) ? attrs_per_CU + remaining_attrs : attrs_per_CU;
    int sum{0}, counter{0};

    // load all elements by thread
    for(int i{attr_start_idx}; i < attr_start_idx + n_attrs; i++) {
        sum     += sums[(attrs_size * cluster * dims) + d * attrs_size + i];
        counter += counts[attrs_size * cluster + i];
    }
    shared_sum[local_idx]   = sum;
    shared_count[local_idx] = counter;

    // tree reduction
    for (int stride = blockDim.x >> 1; stride > 0; stride >>= 1) {
        __syncthreads();
        if (local_idx < stride) {
            shared_sum[local_idx]   += shared_sum[local_idx + stride];
            shared_count[local_idx] += shared_count[local_idx + stride];
        }
    }

    if (local_idx == 0) {
        sums[(attrs_size * cluster * dims) + d * attrs_size] = shared_sum[0];
        counts[attrs_size * cluster]                         = shared_count[0];
    }
}


__global__ void compute_mean(int attrs_size, int dims, float* __restrict__ mean, 
    float* __restrict__ sum, unsigned int* __restrict__ counts)
{
    const int global_index = blockIdx.x * blockDim.x + threadIdx.x;
    const int count        = (1 < counts[attrs_size * global_index]) ? counts[attrs_size * global_index] : 1;
    for(int d{0}; d < dims; d++) {
        int id = (global_index * attrs_size * dims) + (d * attrs_size);
        mean[global_index * dims + d] = sum[id] / count;
    }
}


Device::Device(int _k, int _dims, int n_attrs, std::vector<float>& h_attrs): k(_k), dims(_dims){
    _select_device();
    
    attributes_size     = n_attrs;
    std::tie (blocks, threads, work_items) = _get_block_threads(attributes_size);

    attributes_size_pad = attributes_size + (work_items - attributes_size);
    attributes_bytes    = attributes_size * dims * sizeof(float);
    mean_bytes          = k * dims * sizeof(float);
    sum_size            = k * dims * attributes_size;
    sum_bytes           = sum_size * sizeof(float);
    count_bytes         = k * attributes_size * sizeof(unsigned int);

    hipMalloc(&attributes, attributes_size_pad * dims * sizeof(float));
    hipMalloc(&mean, mean_bytes);
    hipMalloc(&sum, sum_bytes);
    hipMalloc(&counts, count_bytes);
    hipMalloc(&assigments, attributes_size_pad * sizeof(int));

    // init pad values
    hipMemset(attributes, 0, attributes_size_pad * dims * sizeof(float));
    _sync();
    hipMemcpy(attributes, h_attrs.data(), attributes_bytes, hipMemcpyHostToDevice);

    //shuffle attributess to random choose attributess
    std::mt19937 rng(std::random_device{}());
    rng.seed(0);
    std::uniform_int_distribution<size_t> indices(0, attributes_size - 1);
    std::vector<float> h_mean;
    for(int i{0}; i < k; i++) {
        int idx = indices(rng);
        for(int j{0}; j < dims; j++)
            h_mean.push_back(h_attrs[idx + j * attributes_size]);
    }

    hipMemcpy(mean, h_mean.data(), mean_bytes, hipMemcpyHostToDevice);
    hipMemset(sum, 0, sum_bytes);
    hipMemset(counts, 0, count_bytes);
    hipMemset(assigments, 0, attributes_size_pad * sizeof(int)); // potential bug: try init to -1
    _sync();
}


Device::~Device() {
	if(attributes != nullptr) hipFree(attributes);
	if(mean != nullptr)       hipFree(mean);
	if(sum != nullptr)        hipFree(sum);
	if(counts != nullptr)     hipFree(counts);
    if(assigments != nullptr) hipFree(assigments);
}


void Device::_select_device() {
    int device_id{0};
    hipGetDeviceProperties(&_gpu_props, device_id);
    std::cout << "Running on \"" << _gpu_props.name << "\" under CUDA." << std::endl;
}


void Device::run_k_means(int iterations) {
    std::chrono::time_point<std::chrono::high_resolution_clock> start, end;
    float t_assign{0}, t_reduction{0}, t_mean{0};

    for (size_t i{0}; i < iterations; ++i) {
        start = std::chrono::high_resolution_clock::now();
        _assign_clusters();
        _sync();
        end = std::chrono::high_resolution_clock::now();
        t_assign += std::chrono::duration_cast<std::chrono::duration<float>>(end - start).count();

        start = std::chrono::high_resolution_clock::now();
        _reduction();
        _sync();
        end = std::chrono::high_resolution_clock::now();
        t_reduction += std::chrono::duration_cast<std::chrono::duration<float>>(end - start).count();

        start = std::chrono::high_resolution_clock::now();
        _compute_mean();
        _sync();
        end = std::chrono::high_resolution_clock::now();
        t_mean += std::chrono::duration_cast<std::chrono::duration<float>>(end - start).count();
    }

    double total = t_assign + t_reduction + t_mean;
    std::cout << std::endl << "Kernel time: " << std::endl
              << "  * Assign Clusters = " << t_assign << " (s) -> " << t_assign/total*100 << "%" << std::endl
              << "  * Reduction       = " << t_reduction << " (s) -> " << t_reduction/total*100 << "%" << std::endl
              << "  * Mean            = " << t_mean << " (s) -> " << t_mean/total*100 << "%" << std::endl;
}


/*
    Case 1) elements <= max_group_size 
            * threads = elements
            * work_items = elements
            * blocks     = 1
    
    Case 2) elements > max_group_size
            * threads = max_group_size
            * work_items = elements + threads - (elements % threads)
            * blocks     = work_items / threads

*/
std::tuple<int,int,int> Device::_get_block_threads(int elements) {
	const int max_group  = _gpu_props.maxThreadsPerBlock;
    const int threads = (elements <= max_group) ? elements : max_group;
    const int work_items = (elements <= max_group) ? elements : elements + threads - (elements % threads);
    const int blocks     = (elements <= max_group) ? 1 : work_items / threads;

    return std::tuple<int,int,int>(blocks, threads, work_items);
}


void Device::_sync() {
    hipDeviceSynchronize();
}


void Device::_assign_clusters() {
    std::tie (blocks, threads, work_items) = _get_block_threads(attributes_size);

    assign_clusters<<<blocks, threads>>>(
        attributes_size,
        k,
        dims,
        attributes,
        mean,
        sum,
        counts,
        assigments
    );
}


void Device::_reduction() {
	const size_t CUs       = THREADS_EU * EUs_SUBSLICE_NVIDIA_PASCAL;
    const int attrs_per_CU = attributes_size / CUs;
    const int remaining    = attributes_size % CUs;
    int shared_size        = CUs * sizeof(float) + CUs * sizeof(unsigned int);
    dim3 blocks(1, k, dims), threads(CUs);
    tree_reduction<<<blocks, threads, shared_size>>>(
        CUs,
        attrs_per_CU,
        attributes_size,
        remaining,
        k,
        dims,
        sum,
        counts
    );
}


void Device::_compute_mean() {
    std::tie (blocks, threads, work_items) = _get_block_threads(k);
    compute_mean<<<blocks, threads>>>(
        attributes_size,
        dims,
        mean,
        sum,
        counts
    );
}


void Device::save_solution(std::vector<float>& h_mean) {
    hipMemcpy(h_mean.data(), mean, mean_bytes, hipMemcpyDeviceToHost);
    _sync();
}
